#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "AminoAcid.h"
#include <iostream>
#include <vector>
#include <fstream>
#include <cmath>
#include <string>
#define _USE_MATH_DEFINES
#define BOLTZ_CONST .0019872041

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
using namespace std;

__global__ void sum4(double** pd_props, int size, double** pd_sums) {
	int i = threadIdx.x;
	while (i<size-3) {
	double sum = 0.0;
		for (int j = 0; j<4; j++) {
			sum =+ *pd_props[i];
		}
		*pd_sums[i] = sum;
		i++;
	}
}

__global__ void calcProp(double** pd_sums, double** pd_finalProps, int temp) {
	int i = threadIdx.x;
	*pd_finalProps[i] = pow(M_E, -1*(*pd_sums[i])/(BOLTZ_CONST*temp));
}

int main(void) {
	fstream dataFile;
	dataFile.open("aminoAcidList.txt", ios::in);

	thrust::host_vector<AminoAcid*> protein;
	string line;

	while (dataFile >> line) {
		protein.push_back(new AminoAcid(line));
	}
	dataFile.close();
	int temperature;
	cout << "What is the temperature?" << endl;
	cin >> temperature;

	thrust::host_vector<double*> props(protein.size());
	for (int i = 0; i<protein.size(); i++) {
		*props[i] = protein[i]->getPropensity();
	}

	thrust::device_vector<double*> d_props;
	double** pd_props = thrust::raw_pointer_cast(d_props.data());
	thrust::device_vector<double*> d_sums;
	double** pd_sums = thrust::raw_pointer_cast(d_sums.data());
	thrust::device_vector<double*> d_finalProps;
	double** pd_finalProps = thrust::raw_pointer_cast(d_finalProps.data());
	int d_size;
	int d_temp;
	
	int tempSize = 5;
	int *size = &tempSize;
	*size = protein.size();
	int propSize = 1;
	for (int k = 2; k<protein.size()-2; k++) {
		propSize += k;
	}

	hipMalloc((void **) &d_props, propSize*sizeof(double));
	hipMalloc((void **) &d_sums, propSize*sizeof(double));
	hipMalloc((void **) &d_size, sizeof(int));
	hipMalloc((void **) &d_temp, sizeof(int));
	hipMalloc((void **) &d_finalProps, propSize*sizeof(double));

	thrust::host_vector<double*> sums(propSize);
	
	hipMemcpy(&d_props, &props, protein.size()*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(&d_size, &size, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(&d_temp, &temperature, sizeof(int), hipMemcpyHostToDevice);

	sum4<<<1, propSize>>>(pd_props, d_size, pd_sums);
	calcProp<<<1, propSize>>>(pd_sums, pd_finalProps, d_temp);
	
	hipMemcpy(&props, &d_props, propSize*sizeof(double), hipMemcpyDeviceToHost);

	for (int i =0; i<props.size(); i++) {
		if (*props[i] > .05) {
			cout << *props[i] << endl;
		}
	}

	hipFree(&d_finalProps);
	hipFree(&d_props);
	hipFree(&d_sums);
	hipFree(&d_size);
	hipFree(&d_temp);

	return 0;
}
